#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <thrust/device_ptr.h>
#include <omp.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include "mgpu.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int main(){
    printf("\n hola mundo \n");
    
    hipSetDevice(0);
    float *NODE_COORDINATE_2D;NODE_COORDINATE_2D=(float*)calloc(N*2,sizeof(float));
    int *DISTANCE_NODE;DISTANCE_NODE=(int*)calloc(N,sizeof(int));
    int *d_DISTANCE_NODE;hipMalloc( (void **) &d_DISTANCE_NODE, (N)*sizeof( int ));
    int *d_NN_LIST_aux;hipMalloc( (void **) &d_NN_LIST_aux, (N)*sizeof( int ));
    int *d_NN_LIST_cl;hipMalloc( (void **) &d_NN_LIST_cl, (N*cl)*sizeof( int ));
    int *NN_LIST_cl;NN_LIST_cl=(int*)calloc(N*c_l,sizeof(int));
    lectura_2(NODE_COORDINATE_2D); //this  could be optimized
    int i,j;

    printf("\n termine de leer");
    
    make_candidate_list(d_NN_LIST_aux,d_DISTANCE_NODE,DISTANCE_NODE,NODE_COORDINATE_2D,NN_LIST_cl); //this could be optimized. 
    
    
    printf("\n termine de ordenar");
    hipMemcpy(d_NN_LIST_cl,NN_LIST_cl,N*cl*sizeof(int),hipMemcpyHostToDevice);


    float alpha=0.3;
    float beta=5;
    float e=0.01;
    
    int *vec_solution;vec_solution= (int* )malloc(N_e*sizeof(int));
    int *vec_iter;vec_iter= (int* )malloc(N_e*sizeof(int)); //vectores para estadistica
    float *vec_warm_up_time;vec_warm_up_time= (float* )malloc(N_e*sizeof(float));
    float prom_time_2=0.0;
    float *vec_iteration_time;vec_iteration_time=(float*)malloc(N_e*sizeof(float));
    memset(vec_solution,0,N_e*sizeof(int));
    float *vec_ant_iteration_time_series;vec_ant_iteration_time_series=(float*)malloc(ITERACION*sizeof(float));

    for (i=0;i<ITERACION;i++)vec_ant_iteration_time_series[i]=0.0;
    int x;
    float elapsed_for_gpu_ant[N_GPU];
    /*
    Run N_e experiments of ant colony
    */
    for (x=0;x<N_e;x++){
        hipEvent_t start_events[N_GPU];
        int *HORMIGAS_COSTO;HORMIGAS_COSTO=(int*)malloc(ITERACION*M*N_GPU*sizeof(int));
        hipEvent_t end_events[N_GPU];
        float prom_time=0.0;
        float tau_mim,tau_max;
        float P_best=0.001,avg=(float)N/2.0;
        //int *d_PREDECESSOR_ROUTE_MGPU[N_GPU],*d_SUCCESSOR_ROUTE_MGPU[N_GPU]; //2 SOLO PARA FOCUSED
        int *d_COST_MGPU[N_GPU];int *d_NN_LIST_CL_MGPU[N_GPU]; //4
        float *d_NODE_COORDINATE_MGPU[N_GPU];  // 5
        int *d_PREDECESSOR_ROUTE_OP_MGPU[N_GPU],*d_SUCCESSOR_ROUTE_OP_MGPU[N_GPU];  //8
        //9
        int *d_NEW_LIST[N_GPU],*d_NEW_LIST_INDX[N_GPU],*d_ROUTE_AUX[N_GPU];
        float *d_HEURISTIC_PHEROMONE_MGPU[N_GPU]; //10
        /* 10 MULTI GPU VARIABLES */
	        
        int *NEW_LIST_GLOBAL,*NEW_LIST_INDX_GLOBAL;
        NEW_LIST_GLOBAL=(int*)malloc(M*N_GPU*(N+1)*sizeof(int));
        NEW_LIST_INDX_GLOBAL=(int*)malloc(M*N_GPU*(N+1)*sizeof(int));
        int *d_BEST_ANT,*d_OPTIMAL_ROUTE,*d_GLOBAL_COST,*d_GLOBAL_NEW_LIST; //4  
        int *d_ROUTE_NN;
        hipMalloc( (void **) &d_ROUTE_NN, N_e*(N+1)*sizeof(int) );//1
        hipSetDevice(0);
        float *d_HEURISTIC_PHEROMONE,*d_PHEROMONE_MATRIX; //6
        /*6 MASTER GPU VARIABLES*/
        float *HEURISTIC_PHEROMONE;
        bool *VISITED_LIST;
        int *OPTIMAL_ROUTE,*GLOBAL_COST,*BEST_ANT;
        int *PREDECESSOR_ROUTE, *SUCCESSOR_ROUTE;
        float *d_NODE_COORDINATE_2D;
        hiprandState *d_state[N_GPU];
        // global route x2 
        printf("\n fijando memoria en gpu \n");
        hipMalloc( (void **) &d_BEST_ANT, N_GPU*M*sizeof(int) );//1
        hipMalloc( (void **) &d_OPTIMAL_ROUTE, (N+1)*sizeof(int) );//2
        hipMalloc( (void **) &d_HEURISTIC_PHEROMONE, N*cl*sizeof(float));//3
        hipMalloc( (void **) &d_PHEROMONE_MATRIX, N*cl*sizeof(float) );//4
        hipMalloc( (void **) &d_GLOBAL_COST, N_GPU*M*sizeof( int ) );//5
        hipMalloc( (void **) &d_GLOBAL_NEW_LIST, N_GPU*(N+1)*M*sizeof( int ) );//6

        hipMalloc( (void **) &d_NODE_COORDINATE_2D, N*2*sizeof(float ) );//6
        hipMemcpy(d_NODE_COORDINATE_2D,NODE_COORDINATE_2D,2*N*sizeof( float ),hipMemcpyHostToDevice);
        printf("\n fijando memoria en cpu \n");
	
	//ENTROPY CALCULATION
	float *PROB_PHERO;PROB_PHERO=(float*)malloc(N_GPU*M*sizeof(float));
	float *ENTROPY_ITERATION;ENTROPY_ITERATION=(float*)malloc(ITERACION);
	float *PROB_MATRIX;PROB_MATRIX=(float*)malloc(N*cl*sizeof(float));
	float *ENTROPY_VECTOR;ENTROPY_VECTOR=(float*)malloc(N*sizeof(float));
	//ENTROPY CALCULATION
        
	OPTIMAL_ROUTE=(int*)malloc((N+1)*sizeof(int));
        PREDECESSOR_ROUTE=(int*)malloc(N*sizeof(int));
        SUCCESSOR_ROUTE=(int*)malloc(N*sizeof(int));
        GLOBAL_COST=(int*)malloc(N_GPU*M*sizeof(int));
        HEURISTIC_PHEROMONE=(float*)malloc(N*cl*sizeof(float));
        //LOCAL_SEARCH_LIST_MGPU=(int*)malloc(N_GPU*M*N*sizeof(int));
        VISITED_LIST=(bool*)malloc(N*sizeof(bool));
        BEST_ANT=(int*)malloc(N_GPU*M*sizeof(int));
        for (i=0;i<N;i++)VISITED_LIST[i]=false; 
        //12
        printf("\n fijando memoria en mgpu \n");
        for (i=0;i<N_GPU;i++){
            hipSetDevice(i);
            hipEventCreate(&start_events[i]);
            hipEventCreate(&end_events[i]);
            hipMalloc( (void **) &d_NODE_COORDINATE_MGPU[i], 2*N*sizeof( float ) );//1
            hipMemcpy(d_NODE_COORDINATE_MGPU[i],NODE_COORDINATE_2D,2*N*sizeof( float ),hipMemcpyHostToDevice);
            hipMalloc( (void **) &d_NEW_LIST[i], M*(N+1)*sizeof( int ) );//2
            hipMalloc( (void **) &d_ROUTE_AUX[i], M*(N+1)*sizeof( int ) );//2
            hipMalloc( (void **) &d_NEW_LIST_INDX[i], M*(N+1)*sizeof( int ) );
            //hipMalloc( (void **) &d_PREDECESSOR_ROUTE_MGPU[i], M*N*sizeof( int ) );//4
            //hipMalloc( (void **) &d_SUCCESSOR_ROUTE_MGPU[i], M*N*sizeof( int ) );//5
            hipMalloc( (void **) &d_COST_MGPU[i], M*sizeof( int ) );//6
            //hipMalloc( (void **) &d_LOCAL_SEARCH_LIST_MGPU[i], M*N*sizeof( int ) );
            hipMalloc( (void **) &d_NN_LIST_CL_MGPU[i], cl*N*sizeof( int ) ); //7
            hipMemcpy(d_NN_LIST_CL_MGPU[i],NN_LIST_cl,(c_l*N)*sizeof(int),hipMemcpyHostToDevice);
            hipMalloc( (void **) &d_PREDECESSOR_ROUTE_OP_MGPU[i], N*sizeof( int ) );//8
            hipMalloc( (void **) &d_SUCCESSOR_ROUTE_OP_MGPU[i], N*sizeof( int ) );//9
            hipMalloc( (void **) &d_HEURISTIC_PHEROMONE_MGPU[i], N*cl*sizeof( float ) );//10
        }
        hipSetDevice(0);
        printf("\n greedy \n");
        int BEST_GLOBAL_SOLUTION=rutainicial(OPTIMAL_ROUTE,NODE_COORDINATE_2D,NEW_LIST_GLOBAL,NEW_LIST_INDX_GLOBAL,NN_LIST_cl);
        
	hipMemcpy(d_OPTIMAL_ROUTE,OPTIMAL_ROUTE, (N+1)*sizeof(int),hipMemcpyHostToDevice);

        float ini_pheromone=(float)BEST_GLOBAL_SOLUTION;
        float p=pow(P_best,1/(float)N);
        
        printf("\n fijando feromona %f \n",(float)1/ini_pheromone);
        fijar_pheromone<<<(N*cl+32-(N*cl%32)),32>>>(d_PHEROMONE_MATRIX,(float)1/ini_pheromone);
        hipMemcpy(HEURISTIC_PHEROMONE,d_PHEROMONE_MATRIX,(c_l*N)*sizeof(float),hipMemcpyDeviceToHost);
	SAVE_PHEROMONE_MATRIX(HEURISTIC_PHEROMONE, 0, x, alpha, beta, e);
        HEURISTIC_PHEROMONE_CALCULATION<<<N,cl>>>(d_NODE_COORDINATE_2D,d_PHEROMONE_MATRIX,
        d_HEURISTIC_PHEROMONE,d_NN_LIST_cl,alpha,beta);


        thrust::device_ptr<int> dev_inx = thrust::device_pointer_cast(d_BEST_ANT); //utilziar thrust para sorting
        thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(d_GLOBAL_COST);      
        
        
        
        hipMemcpy(HEURISTIC_PHEROMONE,d_HEURISTIC_PHEROMONE,(N*c_l)*sizeof(float),hipMemcpyDeviceToHost);
        printf("\n copiando predecesor\n");


        
        for (i=0;i<N;i++){
            PREDECESSOR_ROUTE[OPTIMAL_ROUTE[i]]=OPTIMAL_ROUTE[i+1]; //seguarda el eje (u,v) osea si tengo rute_predecessor[k]=j quiere decir que en la ruta optima la ciudad k esta conectada con j
            if (i!=0){
                SUCCESSOR_ROUTE[OPTIMAL_ROUTE[i]]=OPTIMAL_ROUTE[i-1];
            }
            else{
                SUCCESSOR_ROUTE[OPTIMAL_ROUTE[i]]=OPTIMAL_ROUTE[N-1];
            }
        }
        for (i=0;i<N_GPU;i++){
            hipSetDevice(i);
            hipMemcpy(d_PREDECESSOR_ROUTE_OP_MGPU[i],PREDECESSOR_ROUTE,(N)*sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy(d_SUCCESSOR_ROUTE_OP_MGPU[i],SUCCESSOR_ROUTE,(N)*sizeof(int),hipMemcpyHostToDevice);
            hipMalloc((void **) &d_state[i], M*sizeof(hiprandState));iniciar_kernel<<<32,M/32>>>(d_state[i],i);
            hipMemcpy(d_HEURISTIC_PHEROMONE_MGPU[i],HEURISTIC_PHEROMONE,N*cl*sizeof(float),hipMemcpyHostToDevice);
            hipDeviceSynchronize();
        }

        int mejor,it,LAST_IMPROVE_ITERATION=0;
        // ITERACIONES 

        int num_gpus;
        hipGetDeviceCount( &num_gpus );
        
        hipSetDevice(0);
        printf("\n INICIANDO ITERACIONES\n");
	float entropy=log2(M);
        for (it=0;it<ITERACION;it++){
            double begin_1 =omp_get_wtime();
            #pragma omp parallel for num_threads(N_GPU)
            for (i=0;i<N_GPU;i++){
                hipSetDevice(i);
                hipEventRecord(start_events[i]);
                LIST_INIT<<<N,min(M,1024)>>>(d_NEW_LIST[i],d_NEW_LIST_INDX[i]);
                ANT_SOLUTION_CONSTRUCT<<<M/4,4>>>(d_HEURISTIC_PHEROMONE_MGPU[i],d_NODE_COORDINATE_MGPU[i],i,d_PREDECESSOR_ROUTE_OP_MGPU[i],
                d_SUCCESSOR_ROUTE_OP_MGPU[i],8,d_state[i],d_NN_LIST_CL_MGPU[i],d_NEW_LIST[i],d_NEW_LIST_INDX[i]);
                /*------------------------ SOBRE ANT_SOLUTION_CONSTRUCT--------------------*/            
                //aumentar el numero de thread ahora parece mejorar el rendimiento
                //en el alg anterior no ocurria eso, cuidado con la memoria compartida
                //por ahora con 4 threads y 4 bloques es lo optimo
                /*------------------------ SOBRE ANT_COST_CALCULATION_LS --------------------*/
                // aumentar el numero de threads mejora el rendimiento  hasta 32 thread
                ANT_COST_CALCULATION_LS<<<M/32,32>>>(d_NEW_LIST[i],d_COST_MGPU[i],d_NODE_COORDINATE_MGPU[i],d_ROUTE_AUX[i],d_state[i]);
                gpuErrchk(hipMemcpyAsync(GLOBAL_COST+i*M,d_COST_MGPU[i],M*sizeof(int),hipMemcpyDeviceToHost));   //esto ahorra 6 ms en 4000 nodos 
                gpuErrchk(hipMemcpyAsync(NEW_LIST_GLOBAL+i*M*(N+1),d_NEW_LIST[i],(N+1)*M*sizeof(int),hipMemcpyDeviceToHost));
                gpuErrchk(hipMemcpyAsync(NEW_LIST_INDX_GLOBAL+i*M*(N+1),d_NEW_LIST_INDX[i],(N+1)*M*sizeof(int),hipMemcpyDeviceToHost));   //esto ahorra 6 ms en 4000 nodos 
                hipDeviceSynchronize();

                hipEventRecord(end_events[i]);

            }
            hipDeviceSynchronize();
            double end_1 =omp_get_wtime();
            hipMemcpy(HORMIGAS_COSTO+it*N_GPU*M,GLOBAL_COST,N_GPU*M*sizeof(int),hipMemcpyHostToHost);

            if(it==0)vec_warm_up_time[x]=(end_1-begin_1)*1000;
            vec_ant_iteration_time_series[it]+=((end_1-begin_1)*1000.0)/((float)N_e);
            
            // printf("\n termino el recorrido en %lf ms\n",(end_1-begin_1)*1000);
            /*
            for(i = 0; i < 4; i++)
                {
                    hipEventElapsedTime(&elapsed_for_gpu_ant[i], start_events[i], end_events[i]);
                    printf("Elapsed time on device %d: %f ms\n", i, elapsed_for_gpu_ant[i]);
                }
            */
            //for (i=0;i<N_GPU;i++){
            //  hipSetDevice(i);
                
            //}
            
            hipSetDevice(0);
            gpuErrchk(hipMemcpy(d_GLOBAL_COST,GLOBAL_COST,N_GPU*M*sizeof(int),hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_GLOBAL_NEW_LIST,NEW_LIST_GLOBAL,N_GPU*(N+1)*M*sizeof(int),hipMemcpyHostToDevice));
            thrust::sequence(thrust::device,dev_inx, dev_inx+N_GPU*M);
            thrust::sort_by_key(thrust::device,dev_ptr, dev_ptr + N_GPU*M, dev_inx,thrust::less<int>());
            hipMemcpy(GLOBAL_COST,d_GLOBAL_COST,N_GPU*M*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(BEST_ANT,d_BEST_ANT,N_GPU*M*sizeof(int),hipMemcpyDeviceToHost);
            
            mejor=BEST_ANT[0];  
            hipSetDevice(0);
            if (it==0){
                GLOBAL_COST[0]=BEST_GLOBAL_SOLUTION;
                tau_max=(float)e*((float)1/(float)BEST_GLOBAL_SOLUTION);
                tau_mim=tau_max*((1-p)/((avg-1)*p));
            }
            if (it%100==0)printf("\n %d\n",GLOBAL_COST[0]);
            if (GLOBAL_COST[0]<BEST_GLOBAL_SOLUTION){
                    
		    BEST_GLOBAL_SOLUTION=GLOBAL_COST[0];
                    LAST_IMPROVE_ITERATION=it;
                    tau_max=(float)e*((float)1/(float)BEST_GLOBAL_SOLUTION);
                    tau_mim=tau_max*((1-p)/((avg-1)*p));
                    if (tau_max<tau_mim)tau_mim=tau_max;
                    printf("\n mejor global = %d en iter= %d en experimento %d con alpha= %f y beta=%f\n",BEST_GLOBAL_SOLUTION,LAST_IMPROVE_ITERATION,x,alpha,beta);
                    //printf("\n tau max =%.16lf \n tau min =%.16lf \n",tau_max,tau_mim);
                    for (i=0;i<N+1;i++){
                        OPTIMAL_ROUTE[i]=NEW_LIST_GLOBAL[mejor*(N+1)+i%N];
                        //printf("%d ",OPTIMAL_ROUTE[i]);
                        if (i>0){
                            PREDECESSOR_ROUTE[OPTIMAL_ROUTE[i-1]]=OPTIMAL_ROUTE[i];
                        }
                        if (i!=0){
                            SUCCESSOR_ROUTE[OPTIMAL_ROUTE[i]]=OPTIMAL_ROUTE[i-1];
                        }
                        else{
                            SUCCESSOR_ROUTE[OPTIMAL_ROUTE[i]]=OPTIMAL_ROUTE[N-1];
                        }
                    }
                    hipMemcpy(d_OPTIMAL_ROUTE,OPTIMAL_ROUTE, (N+1)*sizeof(int),hipMemcpyHostToDevice);
                    for (i=0;i<N_GPU;i++){
                        hipSetDevice(i);
                        hipMemcpy(d_SUCCESSOR_ROUTE_OP_MGPU[i],SUCCESSOR_ROUTE, N*sizeof(int),hipMemcpyHostToDevice);
                        hipMemcpy(d_PREDECESSOR_ROUTE_OP_MGPU[i],PREDECESSOR_ROUTE, N*sizeof(int),hipMemcpyHostToDevice);
                    }
                    hipSetDevice(0);
            }
            
	    float c_1=first_metric(GLOBAL_COST);
            float c_2=second_metric(GLOBAL_COST,BEST_GLOBAL_SOLUTION);
            save_c1_and_c2(c_1,c_2,it,x);
            hipSetDevice(0);
	    EVAPORATION<<<((N*cl+32-(N*cl%32)))/32,32>>>(d_PHEROMONE_MATRIX,e);
	    /*-----------------------ANT SYSTEM--------------------------------*/
	   // PHEROMONE_UPDATE_AS<<<((N+32-(N%32)))/32,32>>>(d_GLOBAL_NEW_LIST,d_BEST_ANT,d_PHEROMONE_MATRIX, 
            //d_NN_LIST_cl,d_GLOBAL_COST,d_OPTIMAL_ROUTE,BEST_GLOBAL_SOLUTION);
            /*-----------------------MMAS      --------------------------------*/
	    /*-----------------------RANK BASED--------------------------------*/
            PHEROMONE_UPDATE<<<((N+32-(N%32)))/32,32>>>(d_GLOBAL_NEW_LIST,d_BEST_ANT,d_PHEROMONE_MATRIX, 
            //d_NN_LIST_cl,d_GLOBAL_COST,d_OPTIMAL_ROUTE,BEST_GLOBAL_SOLUTION);
            /*-----------------------MMAS      --------------------------------*/
            //PHEROMONE_CHECK_MMAS<<<((N*cl+32-(N*cl%32)))/32,32>>>(d_PHEROMONE_MATRIX, tau_max, tau_mim);
            //PHEROMONE_UPDATE<<<((N+32-(N%32)))/32,32>>>(d_GLOBAL_NEW_LIST,d_BEST_ANT,d_PHEROMONE_MATRIX,
            //d_NN_LIST_cl,d_GLOBAL_COST,d_OPTIMAL_ROUTE,BEST_GLOBAL_SOLUTION);
            //PHEROMONE_UPDATE_MMAS<<<((N+32-(N%32)))/32,32>>>(d_GLOBAL_NEW_LIST,d_BEST_ANT,d_PHEROMONE_MATRIX,
            d_NN_LIST_cl,d_GLOBAL_COST,d_OPTIMAL_ROUTE,BEST_GLOBAL_SOLUTION);
            //PHEROMONE_CHECK_MMAS<<<((N*cl+32-(N*cl%32)))/32,32>>>(d_PHEROMONE_MATRIX, tau_max, tau_mim);
            hipMemcpy(HEURISTIC_PHEROMONE,d_PHEROMONE_MATRIX,cl*N*sizeof(float),hipMemcpyDeviceToHost);        
	    shannon_entropy_pheromone(HEURISTIC_PHEROMONE,PROB_MATRIX,ENTROPY_VECTOR);           

            HEURISTIC_PHEROMONE_CALCULATION<<<N,cl>>>(d_NODE_COORDINATE_2D,d_PHEROMONE_MATRIX,
            d_HEURISTIC_PHEROMONE,d_NN_LIST_cl,alpha,beta);
            
            
            hipMemcpy(HEURISTIC_PHEROMONE,d_HEURISTIC_PHEROMONE,(N*c_l)*sizeof(float),hipMemcpyDeviceToHost);
//segmentation	    //entropy=shannon_entropy_p_r(HEURISTIC_PHEROMONE,NEW_LIST_GLOBAL,NN_LIST_cl,PROB_PHERO,entropy,ENTROPY_ITERATION,it);

            for (i=0;i<N_GPU;i++){
                hipSetDevice(i);
                hipMemcpy(d_HEURISTIC_PHEROMONE_MGPU[i],HEURISTIC_PHEROMONE,(N*c_l)*sizeof(float),hipMemcpyHostToDevice);
            }
            hipDeviceSynchronize();
            float end_2 =omp_get_wtime(); 
            prom_time+=(end_2-begin_1)*1000;
        }
        vec_solution[x]=BEST_GLOBAL_SOLUTION;
        printf("\n -------------------------------\n");
        for (i=0;i<N+1;i++)printf("%d ", OPTIMAL_ROUTE[i]);
        printf("\n -------------------------------\n");
        hipSetDevice(0);
        hipFree(d_ROUTE_NN);free(BEST_ANT);hipFree(d_GLOBAL_NEW_LIST);
        hipFree(d_NODE_COORDINATE_2D);
        hipFree(d_BEST_ANT);hipFree(d_OPTIMAL_ROUTE);hipFree(d_GLOBAL_COST);
        hipFree(d_HEURISTIC_PHEROMONE);hipFree(d_PHEROMONE_MATRIX);
        for (i=0;i<N_GPU;i++){
            hipSetDevice(i);hipFree(d_NODE_COORDINATE_MGPU[i]);
            //hipFree(d_PREDECESSOR_ROUTE_MGPU[i]);
            //hipFree(d_SUCCESSOR_ROUTE_MGPU[i]);
            hipFree(d_ROUTE_AUX[i]);
            hipFree(d_COST_MGPU[i]);
            hipFree(d_state[i]);
            hipFree(d_NEW_LIST[i]);
            hipFree(d_NEW_LIST_INDX[i]);
            //hipFree(d_LOCAL_SEARCH_LIST_MGPU[i]);
            hipFree(d_NN_LIST_CL_MGPU[i]);
            hipFree(d_PREDECESSOR_ROUTE_OP_MGPU[i]);
            hipFree(d_SUCCESSOR_ROUTE_OP_MGPU[i]);
            hipFree(d_HEURISTIC_PHEROMONE_MGPU[i]);
            hipEventDestroy(start_events[i]);
            hipEventDestroy(end_events[i]);
        }
        //free(LOCAL_SEARCH_LIST_MGPU);
        escribir_costo(HORMIGAS_COSTO,x);
        free(HORMIGAS_COSTO);
	free(PROB_PHERO);
        free(ENTROPY_ITERATION);
	free(PROB_MATRIX);
	free(ENTROPY_VECTOR);
	free(OPTIMAL_ROUTE);free(VISITED_LIST);
        free(SUCCESSOR_ROUTE);free(PREDECESSOR_ROUTE);free(GLOBAL_COST);
        free(HEURISTIC_PHEROMONE);free(NEW_LIST_GLOBAL);free(NEW_LIST_INDX_GLOBAL);
        prom_time/=(ITERACION);
        prom_time_2+=prom_time;
        vec_iteration_time[x]=prom_time;
        
    }
    prom_time_2/=(N_e);   
    guardar_resultados(vec_warm_up_time,vec_solution,vec_ant_iteration_time_series,vec_iteration_time,alpha,beta,e);
    printf("\n el tiempo promedio es de %f\n ",prom_time_2);
    free(NODE_COORDINATE_2D);free(DISTANCE_NODE);
    hipFree(d_DISTANCE_NODE);free(NN_LIST_cl);hipFree(d_NN_LIST_aux);hipFree(d_NN_LIST_cl);
    return 0;
}
